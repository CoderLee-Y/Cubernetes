#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#define N 8
#define IDX2C(i, j, ld) (((j) * (ld)) + (i))
#define BLOCK_SIZE 8

__global__ void matrixMulSharedKernel_op1(float* fpMatrixA, float* fpMatrixB,
    float* fpMatrixC, int m, int n, int k)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float fCVal = 0.0f;

    __shared__ float shTileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shTileB[BLOCK_SIZE][BLOCK_SIZE];

    int nIter = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int i = 0; i < nIter; i++) {
        // load data from global memory to shared memory
        shTileA[threadIdx.y][threadIdx.x] = fpMatrixA[nRow * k + i * BLOCK_SIZE + threadIdx.x];
        shTileB[threadIdx.y][threadIdx.x] = fpMatrixB[(i * BLOCK_SIZE + threadIdx.y) * n + nCol];

        // sync to wait for all threads in one block to finish loading datas
        __syncthreads();

        // sub-matrix multiply
        for (int l = 0; l < BLOCK_SIZE; l++) {
            fCVal += shTileA[threadIdx.y][l] * shTileB[l][threadIdx.x];
        }

        // sync to wait for all threads in one block to finish compute
        __syncthreads();
    }

    // store results into global memory
    fpMatrixC[nRow * n + nCol] = fCVal;
}

void printfMatrix(float* a, int m, int n)
{
    for (int j = 0; j < m; j++) {
        printf("[");
        for (int i = 0; i < n; i++) {
            printf("\t%lg", a[IDX2C(i, j, m)]);
        }
        printf("\t]\n");
    }
}

int main(void)
{
    int i, j;
    float* devPtrA;
    float* devPtrB;
    float* devPtrC;
    float* a = 0;
    float* b = 0;
    float* c = 0;
    a = (float*)malloc(N * N * sizeof(*a));
    b = (float*)malloc(N * N * sizeof(*b));
    c = (float*)malloc(N * N * sizeof(*b));

    for (j = 0; j < N; j++) {
        for (i = 0; i < N; i++) {
            a[IDX2C(i, j, N)] = (float)(i * N + j + 1);
            b[IDX2C(i, j, N)] = (float)(j * N + i + 1);
        }
    }
    hipMalloc((void**)&devPtrA, N * N * sizeof(*a));
    hipMalloc((void**)&devPtrB, N * N * sizeof(*b));
    hipMalloc((void**)&devPtrC, N * N * sizeof(*c));

    hipMemcpy(devPtrA, a, sizeof(*a) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(devPtrB, b, sizeof(*b) * N * N, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((int)ceil(N / BLOCK_SIZE), (int)ceil(N / BLOCK_SIZE));
    matrixMulSharedKernel_op1<<<dimGrid, dimBlock>>>(devPtrA, devPtrB, devPtrC, N, N, N);

    hipMemcpy(c, devPtrC, sizeof(*c) * N * N, hipMemcpyDeviceToHost);

    printf("A x B = C\n");
    printf("\nA:\n");
    printfMatrix(a, N, N);
    printf("\nB:\n");
    printfMatrix(b, N, N);
    printf("\nC:\n");
    printfMatrix(c, N, N);

    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);

    free(a);
    free(b);
    free(c);
    return EXIT_SUCCESS;
}
