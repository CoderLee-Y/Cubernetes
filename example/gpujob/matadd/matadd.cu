#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#define N 8
#define IDX2C(i, j, ld) (((j) * (ld)) + (i))
#define BLOCK_SIZE 8

__global__ void matrixAdd(const float** A, const float** B, float** C,
    int M, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
        C[i][j] = A[i][j] + B[i][j];
}

void printfMatrix(float* a, int m, int n)
{
    for (int j = 0; j < m; j++) {
        printf("[");
        for (int i = 0; i < n; i++) {
            printf("\t%lg", a[IDX2C(i, j, m)]);
        }
        printf("\t]\n");
    }
}

int main(void)
{
    int i, j;
    float* devPtrA;
    float* devPtrB;
    float* devPtrC;
    float* a = 0;
    float* b = 0;
    float* c = 0;
    a = (float*)malloc(N * N * sizeof(*a));
    b = (float*)malloc(N * N * sizeof(*b));
    c = (float*)malloc(N * N * sizeof(*b));

    for (j = 0; j < N; j++) {
        for (i = 0; i < N; i++) {
            a[IDX2C(i, j, N)] = (float)(i * N + j + 1);
            b[IDX2C(i, j, N)] = (float)(j * N + i + 1);
        }
    }
    hipMalloc((void**)&devPtrA, N * N * sizeof(*a));
    hipMalloc((void**)&devPtrB, N * N * sizeof(*b));
    hipMalloc((void**)&devPtrC, N * N * sizeof(*c));

    hipMemcpy(devPtrA, a, sizeof(*a) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(devPtrB, b, sizeof(*b) * N * N, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((int)ceil(N / BLOCK_SIZE), (int)ceil(N / BLOCK_SIZE));
    matrixAdd<<<dimGrid, dimBlock>>>(devPtrA, devPtrB, devPtrC, N, N);

    hipMemcpy(c, devPtrC, sizeof(*c) * N * N, hipMemcpyDeviceToHost);

    printf("A x B = C\n");
    printf("\nA:\n");
    printfMatrix(a, N, N);
    printf("\nB:\n");
    printfMatrix(b, N, N);
    printf("\nC:\n");
    printfMatrix(c, N, N);

    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);

    free(a);
    free(b);
    free(c);
    return EXIT_SUCCESS;
}
